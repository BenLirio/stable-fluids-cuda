#include "hip/hip_runtime.h"
#include "gold_project.h"
#include "compile_options.h"
#include "macros.h"
#include "idx2.h"
#include "vec2.h"


void gold_project(float *x_velocities, float *y_velocities, float *pressure, float *divergence) {
  float h = 1.0f / sqrt(N);
  for (int y = 1; y <= HEIGHT; y++) {
    for (int x = 1; x <= WIDTH; x++) {
      idx2 idx = idx2(x, y);
      float x_velocity_derivative = x_velocities[IDX2(idx2_wrap(idx2(x+1, y)))] - x_velocities[IDX2(idx2_wrap(idx2(x-1, y)))];
      float y_velocity_derivative = y_velocities[IDX2(idx2_wrap(idx2(x, y+1)))] - y_velocities[IDX2(idx2_wrap(idx2(x, y-1)))];
      divergence[IDX2(idx)] = -0.5f * h * (x_velocity_derivative + y_velocity_derivative);
      pressure[IDX2(idx)] = 0;
    }
  }

  for (int k = 0; k < GAUSS_SEIDEL_ITERATIONS; k++) {
    for (int y = 1; y <= HEIGHT; y++) {
      for (int x = 1; x <= WIDTH; x++) {
        idx2 idx = idx2(x, y);
        float sum = 0;
        for (int i = 0; i < NUM_NEIGHBORS; i++) {
          sum += pressure[IDX2(idx2_wrap(idx2(
            idx.x + adjancent_offsets[i].x,
            idx.y + adjancent_offsets[i].y
          )))];
        }
        pressure[IDX2(idx)] = (divergence[IDX2(idx)] + sum) / 4;
      }
    }
  }

  for (int y = 1; y <= HEIGHT; y++) {
    for (int x = 1; x <= WIDTH; x++) {
      idx2 idx = idx2(x, y);
      x_velocities[IDX2(idx)] -= 0.5f * (pressure[IDX2(idx2_wrap(idx2(x+1, y)))] - pressure[IDX2(idx2_wrap(idx2(x-1, y)))]) / h;
      y_velocities[IDX2(idx)] -= 0.5f * (pressure[IDX2(idx2_wrap(idx2(x, y+1)))] - pressure[IDX2(idx2_wrap(idx2(x, y-1)))]) / h;
    }
  }
}