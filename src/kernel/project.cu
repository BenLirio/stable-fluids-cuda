#include "hip/hip_runtime.h"
#include <kernel/advect.cuh>
#include <util/macros.h>
#include <util/vec2.cuh>
#include <util/idx2.cuh>
#include <util/type_casting.cuh>
#include <util/derivative.cuh>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <kernel/solve.cuh>
#include <gold/solve.cuh>


// __global__ void kernel_project_solve_red_black_naive(float *x_velocities, float *y_velocities, float *pressures, float *divergences, int red) {
//   idx2 idx = idx2(
//     blockIdx.x*blockDim.x + threadIdx.x + 1,
//     blockIdx.y*blockDim.y + threadIdx.y + 1
//   );
//   if (idx.x > WIDTH || idx.y > HEIGHT) return;
//   if (idx.x % 2 == (idx.y + red) % 2) return;
//   pressures[IDX2(idx)] = (divergences[IDX2(idx)] + (
//       pressures[IDX2(idx2_add(idx, idx2(0, 1)))] +
//       pressures[IDX2(idx2_add(idx, idx2(0, -1)))] +
//       pressures[IDX2(idx2_add(idx, idx2(1, 0)))] +
//       pressures[IDX2(idx2_add(idx, idx2(-1, 0)))]
//   )) / 4;
// }

// __global__ void kernel_project_solve_red_black_shared(float *x_velocities, float *y_velocities, float *pressures, float *divergences, int red) {
//   __shared__ float shared_pressures[BLOCK_SIZE+2][BLOCK_SIZE+2];
//   idx2 idx = idx2(
//     blockIdx.x*blockDim.x + threadIdx.x + 1,
//     blockIdx.y*blockDim.y + threadIdx.y + 1
//   );
//   int x = threadIdx.x + 1;
//   int y = threadIdx.y + 1;
//   if (idx.x > WIDTH || idx.y > HEIGHT) return;
//   float divergence;

//   if (idx.x % 2 == (idx.y + red) % 2) {
//     shared_pressures[y+0][x+0] = pressures[IDX2(idx)];
//     return;
//   } else {
//     divergence = divergences[IDX2(idx)];
//     if (x == 1)           shared_pressures[y+0][x-1] = pressures[IDX2(idx2_add(idx, idx2(-1, +0)))];
//     if (x == BLOCK_SIZE)  shared_pressures[y+0][x+1] = pressures[IDX2(idx2_add(idx, idx2(+1, +0)))];
//     if (y == 1)           shared_pressures[y-1][x+0] = pressures[IDX2(idx2_add(idx, idx2(+0, -1)))];
//     if (y == BLOCK_SIZE)  shared_pressures[y+1][x+0] = pressures[IDX2(idx2_add(idx, idx2(+0, +1)))];
//   }

//   __syncthreads();
//   pressures[IDX2(idx)] = (divergence + (
//     shared_pressures[y+0][x+1] +
//     shared_pressures[y+0][x-1] +
//     shared_pressures[y+1][x+0] +
//     shared_pressures[y-1][x+0]
//   )) / 4;
// }

__global__ void kernel_project_prepare(float *x_velocities, float *y_velocities, float *pressures, float *divergences) {
  float h = 1.0f / sqrt((float)N);
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  float x_velocity_derivative = get_x_derivative(x_velocities, idx);
  float y_velocity_derivative = get_y_derivative(y_velocities, idx);
  divergences[IDX2(idx)] = -h * (x_velocity_derivative + y_velocity_derivative)/2;
  pressures[IDX2(idx)] = 0.0f;
}

__global__ void kernel_project_write(float *x_velocities, float *y_velocities, float *pressures, float *divergences) {
  float h = 1.0f / sqrt((float)N);
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  x_velocities[IDX2(idx)] -= get_x_derivative(pressures, idx) / (2*h);
  y_velocities[IDX2(idx)] -= get_y_derivative(pressures, idx) / (2*h);
}

void kernel_project_wrapper(int step, float *x_velocities, float *y_velocities, float *pressures, float *divergences) {
  kernel_project_prepare<<<GRID_DIM, BLOCK_DIM>>>(x_velocities, y_velocities, pressures, divergences);

  float factor = 1.0f;
  float divisor = 4.0f;
  if (OUTPUT&OUTPUT_SOLVE_ERROR) {
    float *expected_values;
    hipMalloc(&expected_values, N*sizeof(float));
    gold_solve_wrapper(expected_values, divergences, pressures, factor, divisor);
    kernel_solve(step, divergences, pressures, expected_values, factor, divisor, PROJECT_TAG);
    hipFree(expected_values);
  } else {
    kernel_solve(step, divergences, pressures, NULL, factor, divisor, PROJECT_TAG);
  }

  kernel_project_write<<<GRID_DIM, BLOCK_DIM>>>(x_velocities, y_velocities, pressures, divergences);
}