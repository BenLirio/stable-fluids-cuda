#include <util/macros.h>
#include <stdio.h>
#include <util/idx2.cuh>
#include <hip/hip_runtime.h>
#include <kernel/solve.cuh>
#include <gold/solve.cuh>
#include <util/state.h>

void kernel_diffuse_wrapper(state_t *state, float *previous_values, float *values, float rate) {
  float factor = TIME_STEP*rate*N;
  float divisor = 1 + 4*factor;
  if (OUTPUT&OUTPUT_SOLVE_ERROR) {
    float *expected_values;
    CUDA_CHECK(hipMalloc(&expected_values, N*sizeof(float)));
    gold_solve_wrapper(expected_values, previous_values, values, factor, divisor);
    kernel_solve(state, previous_values, values, expected_values, factor, divisor, DIFFUSE_TAG);
    hipFree(expected_values);
  } else {
    kernel_solve(state, previous_values, values, NULL, factor, divisor, DIFFUSE_TAG);
  }
}