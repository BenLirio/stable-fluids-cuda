#include "hip/hip_runtime.h"
#include <util/macros.h>
#include <stdio.h>
#include <util/idx2.cuh>
#include <hip/hip_runtime.h>

__global__ void kernel_diffuse_red_black_naive(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;
  if (idx.x % 2 == (idx.y+red) % 2) return;
  values[IDX2(idx)] = (
    previous_values[IDX2(idx)] +
    factor*(
      values[IDX2(idx2_add(idx, idx2(1, 0)))] +
      values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
      values[IDX2(idx2_add(idx, idx2(0, 1)))] +
      values[IDX2(idx2_add(idx, idx2(0, -1)))]
    )
  ) / (1 + 4*factor);
}


__global__ void kernel_diffuse_red_black_shared(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  __shared__ float shared_values[BLOCK_SIZE+2][BLOCK_SIZE+2];

  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;

  int x = threadIdx.x+1;
  int y = threadIdx.y+1;


  float previous_value;
  if (idx.x % 2 == (idx.y+red) % 2) {
    shared_values[x+0][y+0] = values[IDX2(idx)];
    return;
  } else {
    previous_value = previous_values[IDX2(idx)];
    if (x == 1)           shared_values[x-1][y+0] = values[IDX2(idx2_add(idx, idx2(-1, +0)))];
    if (x == BLOCK_SIZE)  shared_values[x+1][y+0] = values[IDX2(idx2_add(idx, idx2(+1, +0)))];
    if (y == 1)           shared_values[x+0][y-1] = values[IDX2(idx2_add(idx, idx2(+0, -1)))];
    if (y == BLOCK_SIZE)  shared_values[x+0][y+1] = values[IDX2(idx2_add(idx, idx2(+0, +1)))];
  }
  __syncthreads();

  values[IDX2(idx)] = (
    previous_value +
    factor*(
      shared_values[x+1][y+0] +
      shared_values[x-1][y+0] +
      shared_values[x+0][y+1] +
      shared_values[x+0][y-1]
    )
  ) / (1 + 4*factor);
}

__global__ void kernel_diffuse_red_black_thread_coarsening(float *previous_values, float *values, float rate, int red) {

  float factor = TIME_STEP*rate*N;
  int coarsening = 2;

  idx2 base_idx = idx2(
    blockIdx.x*(blockDim.x*coarsening) + (threadIdx.x*coarsening) + 1,
    blockIdx.y*(blockDim.y*coarsening) + (threadIdx.y*coarsening) + 1
  );

  for (int y = 0; y < coarsening; y++) {
    for (int x = 0; x < coarsening; x++) {
      idx2 idx = idx2_add(base_idx, idx2(x, y));
      if (idx.x > WIDTH || idx.y > HEIGHT) continue;
      if (idx.x % 2 == (idx.y+red) % 2) continue;
      values[IDX2(idx)] = (
        previous_values[IDX2(idx)] +
        factor*(
          values[IDX2(idx2_add(idx, idx2(1, 0)))] +
          values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
          values[IDX2(idx2_add(idx, idx2(0, 1)))] +
          values[IDX2(idx2_add(idx, idx2(0, -1)))]
        )
      ) / (1 + 4*factor);
    }
  }
}

__global__ void kernel_diffuse_red_black_row_coarsening(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  idx2 base_idx = idx2(blockIdx.x*BLOCK_SIZE*BLOCK_SIZE, blockIdx.y*blockDim.y + threadIdx.y + 1);
  if (idx.y > HEIGHT) return;
  for (int i = 1; i <= BLOCK_SIZE*BLOCK_SIZE; i++) {
    idx2 idx = idx2_add(base_idx, idx2(i, 0));


    if (idx.x > WIDTH || idx.y > HEIGHT) continue;
    if (idx.x % 2 == (idx.y+red) % 2) continue;

    values[IDX2(idx)] = (
      previous_values[IDX2(idx)] +
      factor*(
        values[IDX2(idx2_add(idx, idx2(1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(0, 1)))] +
        values[IDX2(idx2_add(idx, idx2(0, -1)))]
      )
    ) / (1 + 4*factor);
  }
}

void kernel_diffuse_wrapper(float *previous_values, float *values, float rate) {

  void (*kernel_diffuse_red_black)(float *previous_values, float *values, float rate, int red) = kernel_diffuse_red_black_naive;
  dim3 grid_dim = dim3(GRID_DIM.x, GRID_DIM.y);
  dim3 block_dim = dim3(BLOCK_DIM.x, BLOCK_DIM.y);

  if (KERNEL_FLAGS&USE_SHARED_MEMORY) {
    kernel_diffuse_red_black = kernel_diffuse_red_black_shared;
  } else if (KERNEL_FLAGS&USE_THREAD_COARSENING) {
    grid_dim.x <<= 1;
    grid_dim.y <<= 1;
    kernel_diffuse_red_black = kernel_diffuse_red_black_thread_coarsening;
  } else if (KERNEL_FLAGS&USE_ROW_COARSENING) {
    grid_dim.x = 1;
    grid_dim.y = BLOCK_SIZE*BLOCK_SIZE;
    kernel_diffuse_red_black = kernel_diffuse_red_black_row_coarsening;
  }

  for (int i = 0; i < GAUSS_SEIDEL_ITERATIONS; i++) {
    kernel_diffuse_red_black<<<grid_dim, block_dim>>>(previous_values, values, rate, RED);
    kernel_diffuse_red_black<<<grid_dim, block_dim>>>(previous_values, values, rate, BLACK);
  }
}









// Broken

__global__ void kernel_diffuse_single_block(float *previous_values, float *values, float rate) {
  float factor = TIME_STEP*rate*N;
  int x = threadIdx.x+1;
  int y = threadIdx.y+1;
  idx2 idx = idx2(x, y);
  for (int k = 0; k < GAUSS_SEIDEL_ITERATIONS; k++) {
    float next_value = (
      previous_values[IDX2(idx)] +
      factor*(
        values[IDX2(idx2_add(idx, idx2(1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(0, 1)))] +
        values[IDX2(idx2_add(idx, idx2(0, -1)))]
      )
    ) / (1 + 4*factor);
    __syncthreads();
    values[IDX2(idx)] = next_value;
    __syncthreads();
  }
}

__global__ void kernel_diffuse_no_optimization(float *previous_values, float *values, float rate) {
  float factor = TIME_STEP*rate*N;
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  for (int k = 0; k < GAUSS_SEIDEL_ITERATIONS; k++) {
    float next_value = (
      previous_values[IDX2(idx)] +
      factor*(
        values[IDX2(idx2_add(idx, idx2(1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(0, 1)))] +
        values[IDX2(idx2_add(idx, idx2(0, -1)))]
      )
    ) / (1 + 4*factor);
    __syncthreads();
    if (idx.x >= 1 && idx.x <= WIDTH && idx.y >= 1 && idx.y <= HEIGHT)
      values[IDX2(idx)] = next_value;
    __syncthreads();
  }
}

void (*kernel_diffuse)(float *previous_values, float *values, float rate) = kernel_diffuse_no_optimization;

void kernel_diffuse_test_harness(float *previous_values, float *values, float rate) {
  float *d_previous_values, *d_values;
  hipMalloc(&d_previous_values, sizeof(float)*N);
  hipMalloc(&d_values, sizeof(float)*N);

  hipMemcpy(d_previous_values, previous_values, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, sizeof(float)*N, hipMemcpyHostToDevice);

  kernel_diffuse<<<1, dim3(WIDTH, HEIGHT)>>>(d_previous_values, d_values, rate);

  hipMemcpy(values, d_values, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(previous_values, d_previous_values, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipFree(d_values);
  hipFree(d_previous_values);
}