#include "hip/hip_runtime.h"
#include <util/compile_options.h>
#include <util/macros.h>
#include <stdio.h>
#include <util/idx2.cuh>
#include <hip/hip_runtime.h>

__global__ void kernel_diffuse_single_block(float *previous_values, float *values, float rate) {
  float factor = TIME_STEP*rate*N;
  int x = threadIdx.x+1;
  int y = threadIdx.y+1;
  idx2 idx = idx2(x, y);
  for (int k = 0; k < GAUSS_SEIDEL_ITERATIONS; k++) {
    float next_value = (
      previous_values[IDX2(idx)] +
      factor*(
        values[IDX2(idx2_add(idx, idx2(1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(0, 1)))] +
        values[IDX2(idx2_add(idx, idx2(0, -1)))]
      )
    ) / (1 + 4*factor);
    __syncthreads();
    values[IDX2(idx)] = next_value;
    __syncthreads();
  }
}

__global__ void kernel_diffuse_no_optimization(float *previous_values, float *values, float rate) {
  float factor = TIME_STEP*rate*N;
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  for (int k = 0; k < GAUSS_SEIDEL_ITERATIONS; k++) {
    float next_value = (
      previous_values[IDX2(idx)] +
      factor*(
        values[IDX2(idx2_add(idx, idx2(1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
        values[IDX2(idx2_add(idx, idx2(0, 1)))] +
        values[IDX2(idx2_add(idx, idx2(0, -1)))]
      )
    ) / (1 + 4*factor);
    __syncthreads();
    if (idx.x >= 1 && idx.x <= WIDTH && idx.y >= 1 && idx.y <= HEIGHT)
      values[IDX2(idx)] = next_value;
    __syncthreads();
  }
}

void (*kernel_diffuse)(float *previous_values, float *values, float rate) = kernel_diffuse_no_optimization;

__global__ void kernel_diffuse_red_black_naive(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;
  if (idx.x % 2 == (idx.y+red) % 2) return;
  values[IDX2(idx)] = (
    previous_values[IDX2(idx)] +
    factor*(
      values[IDX2(idx2_add(idx, idx2(1, 0)))] +
      values[IDX2(idx2_add(idx, idx2(-1, 0)))] +
      values[IDX2(idx2_add(idx, idx2(0, 1)))] +
      values[IDX2(idx2_add(idx, idx2(0, -1)))]
    )
  ) / (1 + 4*factor);
}

__global__ void kernel_diffuse_red_black_shared(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  __shared__ float shared_values[BLOCK_SIZE+2][BLOCK_SIZE+2];

  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;

  int x = threadIdx.x+1;
  int y = threadIdx.y+1;

                        shared_values[x+0][y+0] = values[IDX2(idx)];
  if (x == 1)           shared_values[x-1][y+0] = values[IDX2(idx2_add(idx, idx2(-1, +0)))];
  if (x == BLOCK_SIZE)  shared_values[x+1][y+0] = values[IDX2(idx2_add(idx, idx2(+1, +0)))];
  if (y == 1)           shared_values[x+0][y-1] = values[IDX2(idx2_add(idx, idx2(+0, -1)))];
  if (y == BLOCK_SIZE)  shared_values[x+0][y+1] = values[IDX2(idx2_add(idx, idx2(+0, +1)))];

  if (idx.x % 2 == (idx.y+red) % 2) return;
  __syncthreads();

  values[IDX2(idx)] = (
    previous_values[IDX2(idx)] +
    factor*(
      shared_values[x+1][y+0] +
      shared_values[x-1][y+0] +
      shared_values[x+0][y+1] +
      shared_values[x+0][y-1]
    )
  ) / (1 + 4*factor);
}

__global__ void kernel_diffuse_red_black_shared_and_neighbor_map(float *previous_values, float *values, float rate, int red) {
  float factor = TIME_STEP*rate*N;
  __shared__ float shared_values[BLOCK_SIZE+2][BLOCK_SIZE+2];

  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;

  int x = threadIdx.x+1;
  int y = threadIdx.y+1;

                        shared_values[x+0][y+0] = values[IDX2(idx)];
  if (x == 1)           shared_values[x-1][y+0] = values[IDX2(idx2_add(idx, idx2(-1, +0)))];
  if (x == BLOCK_SIZE)  shared_values[x+1][y+0] = values[IDX2(idx2_add(idx, idx2(+1, +0)))];
  if (y == 1)           shared_values[x+0][y-1] = values[IDX2(idx2_add(idx, idx2(+0, -1)))];
  if (y == BLOCK_SIZE)  shared_values[x+0][y+1] = values[IDX2(idx2_add(idx, idx2(+0, +1)))];

  if (idx.x % 2 == (idx.y+red) % 2) return;
  __syncthreads();

  values[IDX2(idx)] = (
    previous_values[IDX2(idx)] +
    factor*(
      shared_values[x+1][y+0] +
      shared_values[x-1][y+0] +
      shared_values[x+0][y+1] +
      shared_values[x+0][y-1]
    )
  ) / (1 + 4*factor);
}

void (*kernel_diffuse_red_black)(float *previous_values, float *values, float rate, int red) = kernel_diffuse_red_black_shared;

void kernel_diffuse_wrapper(float *previous_values, float *values, float rate) {
  for (int i = 0; i < GAUSS_SEIDEL_ITERATIONS; i++) {
    kernel_diffuse_red_black<<<GRID_DIM, BLOCK_DIM>>>(previous_values, values, rate, RED);
    kernel_diffuse_red_black<<<GRID_DIM, BLOCK_DIM>>>(previous_values, values, rate, BLACK);
  }
}

void kernel_diffuse_test_harness(float *previous_values, float *values, float rate) {
  float *d_previous_values, *d_values;
  hipMalloc(&d_previous_values, sizeof(float)*N);
  hipMalloc(&d_values, sizeof(float)*N);

  hipMemcpy(d_previous_values, previous_values, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, sizeof(float)*N, hipMemcpyHostToDevice);

  kernel_diffuse<<<1, dim3(WIDTH, HEIGHT)>>>(d_previous_values, d_values, rate);

  hipMemcpy(values, d_values, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(previous_values, d_previous_values, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipFree(d_values);
  hipFree(d_previous_values);
}