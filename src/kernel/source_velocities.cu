#include "hip/hip_runtime.h"
#include <kernel/source_velocities.cuh>
#include <util/vec2.cuh>
#include <util/macros.h>
#include <util/idx2.cuh>

#include <util/type_casting.cuh>
#include <hip/hip_runtime.h>

__global__ void kernel_source_velocities_single_block(float *previous_x_velocities, float *previous_y_velocities, float *x_velocities, float *y_velocities, int current_step) {
  idx2 idx = idx2(
    threadIdx.x + 1,
    threadIdx.y + 1
  );

  vec2 center = vec2((WIDTH/2.0)+0.5, (HEIGHT/2.0)+0.5);
  vec2 position = vec2_of_idx2(idx);
  float distance = vec2_scaled_dist(center, position);
  float magnitude = 1.0/(distance*distance);
  float percent_complete = (float)current_step / (float)NUM_STEPS;
  float x_magnitude = magnitude*cos(percent_complete*M_PI*VELOCITY_SPIN_RATE);
  float y_magnitude = magnitude*sin(percent_complete*M_PI*VELOCITY_SPIN_RATE);

  x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  previous_x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  previous_y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
}

__global__ void kernel_source_velocities_no_optimization(float *previous_x_velocities, float *previous_y_velocities, float *x_velocities, float *y_velocities, int current_step) {
  idx2 idx = idx2(
    blockIdx.x*blockDim.x + threadIdx.x + 1,
    blockIdx.y*blockDim.y + threadIdx.y + 1
  );
  if (idx.x > WIDTH || idx.y > HEIGHT) return;

  vec2 center = vec2((WIDTH/2.0)+0.5, (HEIGHT/2.0)+0.5);
  vec2 position = vec2_of_idx2(idx);
  float distance = vec2_scaled_dist(center, position);
  float magnitude = 1.0/(distance*distance);
  float x_magnitude = magnitude*cos(current_step*TIME_STEP*M_PI*VELOCITY_SPIN_RATE);
  float y_magnitude = magnitude*sin(current_step*TIME_STEP*M_PI*VELOCITY_SPIN_RATE);
  

  x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  previous_x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
  previous_y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP*VELOCITY_SOURCE_MAGNITUDE;
}

void (*kernel_source_velocities)(float *previous_x_velocities, float *previous_y_velocities, float *x_velocities, float *y_velocities, int current_step) = kernel_source_velocities_no_optimization;

void kernel_source_velocities_wrapper(float *previous_x_velocities, float *previous_y_velocities, float *x_velocities, float *y_velocities, int current_step) {
  size_t number_of_bytes = sizeof(float)*N;

  float *device_previous_x_velocities;
  float *device_x_velocities;
  float *device_previous_y_velocities;
  float *device_y_velocities;

  hipMalloc(&device_previous_x_velocities, number_of_bytes);
  hipMalloc(&device_x_velocities, number_of_bytes);
  hipMalloc(&device_previous_y_velocities, number_of_bytes);
  hipMalloc(&device_y_velocities, number_of_bytes);

  hipMemcpy(device_previous_x_velocities, previous_x_velocities, number_of_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_x_velocities, x_velocities, number_of_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_previous_y_velocities, previous_y_velocities, number_of_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_y_velocities, y_velocities, number_of_bytes, hipMemcpyHostToDevice);

  kernel_source_velocities<<<1, dim3(WIDTH, HEIGHT)>>>(device_previous_x_velocities, device_previous_y_velocities, device_x_velocities, device_y_velocities, current_step);

  hipMemcpy(previous_x_velocities, device_previous_x_velocities, number_of_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(x_velocities, device_x_velocities, number_of_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(previous_y_velocities, device_previous_y_velocities, number_of_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(y_velocities, device_y_velocities, number_of_bytes, hipMemcpyDeviceToHost);

  hipFree(device_previous_x_velocities);
  hipFree(device_x_velocities);
  hipFree(device_previous_y_velocities);
  hipFree(device_y_velocities);
}