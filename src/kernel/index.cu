#include <hip/hip_runtime.h>
#include <util/compile_options.h>
#include <kernel/diffuse.cuh>
#include <kernel/advect.cuh>
#include <kernel/project.cuh>
#include <util/macros.h>
#include <kernel/index.cuh>
#include <kernel/source_colors.cuh>
#include <kernel/source_velocities.cuh>
#include <kernel/sink_velocities.cuh>
#include <kernel/sink_colors.cuh>
#include <util/state.h>
#include <stdio.h>

void kernel_step(state_t state, int current_step) {
  state_property_t *c = state.colors;
  state_property_t *x = state.x_velocities;
  state_property_t *y = state.y_velocities;
  state_property_t *p = state.pressures;
  state_property_t *d = state.divergences;

  if (USE_SOURCE_COLORS)
    kernel_source_colors<<<1, dim3(WIDTH, HEIGHT)>>>(c->previous, c->current);
  if (USE_SINK_COLORS)
    kernel_sink_colors<<<1, dim3(WIDTH, HEIGHT)>>>(c->previous, c->current);
  if (USE_DENSITY_DIFFUSE) {
    state_property_step(c);
    kernel_diffuse<<<1, dim3(WIDTH, HEIGHT)>>>(c->previous, c->current, DIFFUSION_RATE);
  }
  if (USE_DENSITY_ADVECT) {
    state_property_step(c);
    kernel_advect<<<1, dim3(WIDTH, HEIGHT)>>>(c->previous, c->current, x->current, y->current);
  }

  if (USE_SOURCE_VELOCITIES)
    kernel_source_velocities<<<1, dim3(WIDTH, HEIGHT)>>>(x->previous, y->previous, x->current, y->current, current_step);
  if (USE_SINK_VELOCITIES)
    kernel_sink_velocities<<<1, dim3(WIDTH, HEIGHT)>>>(x->previous, y->previous, x->current, y->current);
  if (USE_VELOCITY_DIFFUSE) {
    state_property_step(x);
    kernel_diffuse<<<1, dim3(WIDTH, HEIGHT)>>>(x->previous, x->current, VISCOSITY);
    state_property_step(y);
    kernel_diffuse<<<1, dim3(WIDTH, HEIGHT)>>>(y->previous, y->current, VISCOSITY);
    kernel_project<<<1, dim3(WIDTH, HEIGHT)>>>(x->current, y->current, p->current, d->current);
  }
  if (USE_VELOCITY_ADVECT) {
    state_property_step(x);
    state_property_step(y);
    kernel_advect<<<1, dim3(WIDTH, HEIGHT)>>>(x->previous, x->current, x->previous, y->previous);
    kernel_advect<<<1, dim3(WIDTH, HEIGHT)>>>(y->previous, y->current, x->previous, y->previous);
    kernel_project<<<1, dim3(WIDTH, HEIGHT)>>>(x->current, y->current, p->current, d->current);
  }
}

void kernel_step_wrapper(state_t state, int current_step) {
  state_t device_state;
  state_cuda_malloc(&device_state);

  hipMemcpy(device_state.colors->current, state.colors->current, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.colors->previous, state.colors->previous, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.x_velocities->current, state.x_velocities->current, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.x_velocities->previous, state.x_velocities->previous, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.y_velocities->current, state.y_velocities->current, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.y_velocities->previous, state.y_velocities->previous, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.pressures->current, state.pressures->current, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.pressures->previous, state.pressures->previous, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.divergences->current, state.divergences->current, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_state.divergences->previous, state.divergences->previous, N*sizeof(float), hipMemcpyHostToDevice);

  kernel_step(device_state, current_step);

  hipMemcpy(state.colors->current, device_state.colors->current, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.colors->previous, device_state.colors->previous, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.x_velocities->current, device_state.x_velocities->current, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.x_velocities->previous, device_state.x_velocities->previous, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.y_velocities->current, device_state.y_velocities->current, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.y_velocities->previous, device_state.y_velocities->previous, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.pressures->current, device_state.pressures->current, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.pressures->previous, device_state.pressures->previous, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.divergences->current, device_state.divergences->current, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(state.divergences->previous, device_state.divergences->previous, N*sizeof(float), hipMemcpyDeviceToHost);

  state_cuda_free(device_state);
}