#include "hip/hip_runtime.h"
#include <kernel/advect.cuh>
#include <util/macros.h>
#include <util/compile_options.h>
#include <util/vec2.cuh>
#include <util/idx2.cuh>
#include <util/type_casting.cuh>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__constant__ idx2 constant_lower_right_square_offsets[NUM_NEIGHBORS];

__global__ void kernel_advect(float *previous_values, float *values, float *x_velocities, float *y_velocities) {
  float alpha = TIME_STEP*sqrt((float)N);
  int x = threadIdx.x + 1;
  int y = threadIdx.y + 1;
  idx2 idx = idx2(x, y);
  vec2 pos = vec2((float)x, (float)y);
  vec2 velocity = vec2(x_velocities[IDX2(idx)], y_velocities[IDX2(idx)]);
  vec2 scaled_velocity = vec2_scale(-alpha, velocity);
  vec2 pos_offset_by_velocity = vec2_add(pos, scaled_velocity);
  idx2 idx_offset_by_velocity = idx2_of_vec2(pos_offset_by_velocity);
  vec2 pos_offset_by_velocity_floored = vec2_of_idx2(idx_offset_by_velocity);
  float wx0 = vec2_x_dist(pos_offset_by_velocity, pos_offset_by_velocity_floored);
  float wx1 = 1 - wx0;
  float wy0 = vec2_y_dist(pos_offset_by_velocity, pos_offset_by_velocity_floored);
  float wy1 = 1 - wy0;
  float weights[NUM_NEIGHBORS] = {
    wx1*wy1,
    wx1*wy0,
    wx0*wy1,
    wx0*wy0,
  };
  float new_value = 0.0;
  for (int i = 0; i < NUM_NEIGHBORS; i++) {
    float weight = weights[i];
    idx2 neighbor_idx = idx2_add(idx_offset_by_velocity, constant_lower_right_square_offsets[i]);
    new_value += weight*previous_values[IDX2(neighbor_idx)];
  }
  values[IDX2(idx)] = new_value;
}

void kernel_advect_wrapper(float *previous_values, float *values, float *x_velocities, float *y_velocities) {
  hipMemcpyToSymbol(HIP_SYMBOL(constant_lower_right_square_offsets), lower_right_square_offsets, sizeof(idx2)*NUM_NEIGHBORS);

  float *d_previous_values, *d_values, *d_x_velocities, *d_y_velocities;
  hipMalloc(&d_previous_values, sizeof(float)*N);
  hipMalloc(&d_values, sizeof(float)*N);
  hipMalloc(&d_x_velocities, sizeof(float)*N);
  hipMalloc(&d_y_velocities, sizeof(float)*N);

  hipMemcpy(d_previous_values, previous_values, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_x_velocities, x_velocities, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_y_velocities, y_velocities, sizeof(float)*N, hipMemcpyHostToDevice);

  kernel_advect<<<1, dim3(WIDTH, HEIGHT)>>>(d_previous_values, d_values, d_x_velocities, d_y_velocities);

  hipMemcpy(values, d_values, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(previous_values, d_previous_values, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(x_velocities, d_x_velocities, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(y_velocities, d_y_velocities, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipFree(d_values);
  hipFree(d_previous_values);
  hipFree(d_x_velocities);
  hipFree(d_y_velocities);
}