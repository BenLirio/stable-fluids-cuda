#include "hip/hip_runtime.h"
#include <gold/source_velocities.cuh>
#include <util/compile_options.h>
#include <util/idx2.cuh>
#include <util/vec2.cuh>
#include <util/state.h>
#include <util/type_casting.cuh>

void gold_source_velocities(float *previous_x_velocities, float *previous_y_velocities, float *x_velocities, float *y_velocities, int current_step) {
  vec2 center = vec2((WIDTH/2.0)+0.5, (HEIGHT/2.0)+0.5);
  for (int y = 1; y <= HEIGHT; y++) {
    for (int x = 1; x <= WIDTH; x++) {
      idx2 idx = idx2(x, y);
      vec2 position = vec2_of_idx2(idx);

      float distance = vec2_scaled_dist(center, position);
      float magnitude = 1/(distance*distance);
      float percent_complete = (float)current_step / (float)NUM_STEPS;
      float x_magnitude = magnitude*cos(percent_complete*M_PI*10.0);
      float y_magnitude = magnitude*sin(percent_complete*M_PI*10.0);

      x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP;
      previous_x_velocities[IDX2(idx)] += x_magnitude*TIME_STEP;
      y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP;
      previous_y_velocities[IDX2(idx)] += y_magnitude*TIME_STEP;
    }
  }
}