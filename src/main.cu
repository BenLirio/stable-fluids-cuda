#include "hip/hip_runtime.h"
#include <gold/index.h>

#include <util/macros.h>
#include <util/state.h>
#include <util/idx2.cuh>
#include <stdio.h>
#include <util/vec2.cuh>
#include <kernel/index.cuh>
#include <omp.h>
#include <stdlib.h>
#include <util/log.cuh>

void output_gif_frame(float *colors, int i) {
  if (i != 0)
    printf(",");
  for (int y = 0; y < HEIGHT; y++) {
    for (int x = 0; x < WIDTH; x++) {
      printf("%f", colors[y*WIDTH+x]);
      if (y != HEIGHT - 1 || x != WIDTH - 1)
        printf(",");
    }
  }
}

int main() {
  int log_id;
  state_t *state = (state_t*)malloc(sizeof(state_t));
  float *colors;

  if (USE_GOLD) {
    state_create(state);
    colors = state->all_colors[0]->cur;
  } else {
    state_cuda_create(state);
    if (OUTPUT&OUTPUT_GIF) colors = (float*)malloc(N*sizeof(float));
  }

  for (state->step = 0; state->step < NUM_STEPS; state->step++) {
    empty_log_buffer(state);

    log_id = log(state, rand(), STEP_TAG);
    state_push(state);
    if (USE_GOLD) gold_step(state);
    else kernel_step(state);
    state_pop(state);
    log(state, log_id, STEP_TAG);

    if (OUTPUT&OUTPUT_GIF) {
      CUDA_CHECK(hipMemcpy(colors, state->all_colors[0]->cur, N*sizeof(float), hipMemcpyDeviceToHost));
      output_gif_frame(colors, state->step);
    }
  }

  if (USE_GOLD) {
    state_destroy(state);
  } else {
    state_cuda_destroy(state);
    if (OUTPUT&OUTPUT_GIF) free(colors);
  }

  return 0;
}