#include "hip/hip_runtime.h"
#include <gold/index.h>

#include <util/macros.h>
#include <util/state.h>
#include <util/idx2.cuh>
#include <stdio.h>
#include <util/vec2.cuh>
#include <kernel/index.cuh>
#include <omp.h>
#include <stdlib.h>
#include <util/log.cuh>

void output_gif_frame(float *colors, int i) {
  if (i != 0)
    printf(",");
  for (int y = 0; y < HEIGHT; y++) {
    for (int x = 0; x < WIDTH; x++) {
      printf("%f", colors[y*WIDTH+x]);
      if (y != HEIGHT - 1 || x != WIDTH - 1)
        printf(",");
    }
  }
}

int main() {
  state_t *state = (state_t*)malloc(sizeof(state_t));
  float *colors;

  if (USE_GOLD) {
    state_create(state);
    colors = state->all_colors[0]->cur;
  } else {
    state_cuda_create(state);
    if (OUTPUT&OUTPUT_GIF) colors = (float*)malloc(N*sizeof(float));
  }

  for (state->step = 0; state->step < NUM_STEPS; state->step++) {
    empty_log_buffer(state);
    if (USE_GOLD) {
      gold_step(state);
    } else {
      kernel_step(state);
      if (OUTPUT&OUTPUT_GIF)
        CUDA_CHECK(hipMemcpy(colors, state->all_colors[0]->cur, N*sizeof(float), hipMemcpyDeviceToHost));
    }
    if (OUTPUT&OUTPUT_GIF)
      output_gif_frame(colors, state->step);
  }

  if (USE_GOLD) {
    state_destroy(state);
  } else {
    state_cuda_destroy(state);
    if (OUTPUT&OUTPUT_GIF) free(colors);
  }

  return 0;
}