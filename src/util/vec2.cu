#include "hip/hip_runtime.h"
#include <util/vec2.cuh>

#include <util/macros.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__device__ __host__ float positive_fmod(float x, float m) {
  float r = fmod(x, m);
  return r < 0 ? r + m : r;
}

__device__ __host__ vec2 vec2_wrap(vec2 u) {
  float offset = 0.5;
  return vec2(
    positive_fmod(u.x-offset, (float)WIDTH) + offset,
    positive_fmod(u.y-offset, (float)HEIGHT) + offset
  );
}

__device__ __host__ vec2 vec2_add(vec2 u, vec2 v) {
  return vec2_wrap(vec2(u.x + v.x, u.y + v.y));
}

__device__ __host__ vec2 vec2_scale(float s, vec2 u) {
  return vec2_wrap(vec2(u.x * s, u.y * s));
}

__device__ __host__ float vec2_x_dist(vec2 u, vec2 v) {
  float dx = fabs(u.x - v.x);
  if (dx > WIDTH - dx) dx = WIDTH - dx;
  return dx;
}

__device__ __host__ float vec2_y_dist(vec2 u, vec2 v) {
  float dy = fabs(u.y - v.y);
  if (dy > HEIGHT - dy) dy = HEIGHT - dy;
  return dy;
}

__device__ __host__ float vec2_dist(vec2 u, vec2 v) {
  float dx = vec2_x_dist(u, v);
  float dy = vec2_y_dist(u, v);
  return sqrt(dx*dx + dy*dy);
}


__device__ __host__ float vec2_scaled_dist(vec2 u, vec2 v) {
  float dx = fabs(u.x - v.x);
  if (dx > WIDTH - dx) dx = WIDTH - dx;
  dx = dx/((float) WIDTH);
  float dy = fabs(u.y - v.y);
  if (dy > HEIGHT - dy) dy = HEIGHT - dy;
  dy = dy/((float) HEIGHT);
  return sqrt(dx*dx + dy*dy);
}